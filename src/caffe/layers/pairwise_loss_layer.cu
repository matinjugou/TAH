#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/pairwise_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ForwardGPU(const int nthreads, const int num, const Dtype* similarity, 
       const Dtype* exp_product, const Dtype* product, const Dtype threshold, Dtype* count, Dtype* loss_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
      count[index] = Dtype(1.0);
      if((threshold >= 0) && (product[index] >= threshold)){
          loss_data[index] = product[index] * (1 - (similarity[index] > 0));
      }
      else{
          loss_data[index] = log(1 + exp_product[index]) - (similarity[index] > 0) * product[index];
      }
  }
}

template <typename Dtype>
__global__ void TSNEProduct(const int nthreads, const int outer_num, const int inner_num, const Dtype* input1, const Dtype* input2, Dtype* tproduct, Dtype* out){
    CUDA_KERNEL_LOOP(index, nthreads){
        int data_id1 = index / outer_num;
        int data_id2 = index % outer_num;
        Dtype sum = 0;
        for (int i = 0; i < inner_num; i++){
            sum += (input1[data_id1 * inner_num + i] - input2[data_id2 * inner_num + i])*(input1[data_id1 * inner_num + i] - input2[data_id2 * inner_num + i]);
        }
        tproduct[index] = sum;
        out[index] = (inner_num / 2) / (1.0 + sum);
    }
}

template <typename Dtype>
void PairwiseLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Dtype* similarity = pairwise_sim_.mutable_gpu_data();
  Dtype* dot_product = pairwise_sim_.mutable_gpu_diff();
  Dtype* exp_product = loss_.mutable_gpu_diff();
  Dtype* loss_data = loss_.mutable_gpu_data();
  Dtype* count = temp_.mutable_gpu_data();
  Dtype* label = bottom[1]->mutable_gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  int nthreads = outer_num_ * outer_num_;
  //calculate similarity matrix according to label
  caffe_gpu_gemm(CblasNoTrans, CblasTrans, outer_num_, outer_num_, label_dim_, 
          Dtype(1.0), label, label, Dtype(0.0), similarity);
  //caffe_gpu_memcpy(bottom[0]->count() * sizeof(Dtype), bottom[0]->gpu_data(), bottom[0]->mutable_cpu_data());
  //caffe_gpu_memcpy(bottom[1]->count() * sizeof(Dtype), bottom[1]->gpu_data(), bottom[1]->mutable_cpu_data());
  //char filename[100];
  //sprintf(filename, "dataout%d.txt", itercount);
  //std::ofstream fout(filename, ios::out);
 // LOG(INFO)<<"size"<<sizeof(bottom[0]->cpu_data()[0]);
  //for (int i = 0; i < bottom[0]->count(); i++)
  //{
  // /   //fout.write((char*)(bottom[0]->cpu_data()[i]), sizeof(bottom[0]->cpu_data()[i]));
  //      fout<<bottom[0]->cpu_data()[i]<<' ';
 //}
  //fout.close();
  //sprintf(filename, "labelout%d.txt", itercount);
  //fout.open(filename, ios::out);
  //for (int i = 0; i < bottom[1]->count(); i++)
  //{
       //fout.write((char*)(bottom[1]->cpu_data()[i]), sizeof(bottom[1]->cpu_data()[i]));
  //      fout<<bottom[1]->cpu_data()[i]<<' ';
   //}
   //fout.close();
  //itercount++;
  if (method == 0){
    // use t-sne
    Dtype* tproduct = tsne_.mutable_gpu_data();
    TSNEProduct<Dtype><<<CAFFE_GET_BLOCKS(nthreads), 
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, outer_num_, inner_num_, bottom_data, bottom_data, tproduct, dot_product);
    //caffe_gpu_scal(nthreads, Dtype(10)/128, dot_product);
    //std::ofstream fout3("sumout.txt", ios::out);
    //caffe_gpu_memcpy(sizeof(Dtype)*tsne_.count(), tsne_.gpu_data(), tsne_.mutable_cpu_data());
    //for (int i = 0; i < tsne_.count(); i++)
    //{
//	fout3<<tsne_.cpu_data()[i]<<" ";
    //}
    //fout3.close();
  }
  else{
    // use dhn
    caffe_gpu_gemm(CblasNoTrans, CblasTrans, outer_num_, outer_num_, inner_num_, 
      Dtype(1.0), bottom_data, bottom_data, Dtype(0.0), dot_product);
  }
  
  //caffe_gpu_memcpy(sizeof(Dtype)*pairwise_sim_.count(), pairwise_sim_.gpu_diff(), pairwise_sim_.mutable_cpu_diff());
  //fout.open("dotproductout.txt", ios::out);
  //for (int i = 0; i < pairwise_sim_.count(); i++)
  //{
       //fout.write((char*)(bottom[1]->cpu_data()[i]), sizeof(bottom[1]->cpu_data()[i]));
   //     fout<<pairwise_sim_.cpu_diff()[i]<<' ';
   //}
   //fout.close();

  caffe_gpu_exp(outer_num_ * outer_num_, dot_product, exp_product);
  
  //calculate pairwise loss
  ForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, outer_num_, similarity, exp_product, 
              dot_product, threshold_, count, loss_data);
  
  Dtype loss, count_num;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  caffe_gpu_asum(nthreads, count, &count_num);
  loss /= (count_num > 0 ? count_num : Dtype(1));
  top[0]->mutable_cpu_data()[0] = loss;
  //caffe_gpu_memcpy(sizeof(Dtype) * loss_.count(), loss_.gpu_data(), loss_.mutable_cpu_data());
  //fout.open("lossout.txt", ios::out);
  //for (int i = 0; i < loss_.count(); i++)
  //{
        //fout.write((char*)(bottom[1]->cpu_data()[i]), sizeof(bottom[1]->cpu_data()[i]));
  //      fout<<loss_.cpu_data()[i]<<' ';
  //}
  //fout.close();

}

template <typename Dtype>
__global__ void BackwardGPU(const int nthreads, const int outer_num, const int inner_num,
          const Dtype* similarity, const Dtype* exp_product,const Dtype* tproduct,const Dtype threshold,  Dtype* count, Dtype* diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
        if ((inner_num / 2)/(1. + tproduct[index])>threshold)
	{
		diff[index] = 1.0 * (1- (similarity[index] > 0));
	}
	else
	{
	diff[index] = (
          1 / (1 + 1 / exp_product[index]) - 
          (similarity[index] > 0)
	);
      }
      diff[index] /= Dtype(outer_num * outer_num);
      //diff[index] *= Dtype(10)/ 128;
      count[index] = Dtype(1.0);
  }
}

template <typename Dtype>
__global__ void TSNEBackward31(const int nthreads, const Dtype scale, const int outer_num, const int inner_num, const Dtype* top_diff, const Dtype* tproduct,const Dtype* similarity, const Dtype threshold,  const Dtype* input, Dtype* diff, Dtype* diff2, Dtype* diff3){
    CUDA_KERNEL_LOOP(index, nthreads){  
	for (int i = 0; i < outer_num; i++)
	{
		Dtype temp_diff = top_diff[int(index / inner_num) * outer_num + i] * (-(inner_num / 2) / ((1.0+tproduct[int(index / inner_num) * outer_num + i])*(1.0+tproduct[int(index / inner_num) * outer_num + i])));
		//diff2[int(index / inner_num) * outer_num + i] = (-(inner_num / 2) / ((1.0+tproduct[int(index / inner_num) * outer_num + i])*(1.0+tproduct[int(index / inner_num) * outer_num + i])));
		diff[index] +=  temp_diff * 2.0 * (input[index] - input[i * inner_num + index % inner_num]);
		//diff[index] += scale * temp_diff * 2.0 * (input[index] - input[i * inner_num + index % inner_num]);	
            	//diff3[index] += 2.0*(input[index] - input[i * inner_num + index % inner_num]);
	}
    }
}  

template <typename Dtype>
__global__ void TSNEBackward32(const int nthreads, const Dtype scale, const int outer_num, const int inner_num, const Dtype* top_diff, const Dtype* tproduct, const Dtype* similarity, const Dtype threshold, const Dtype* input, Dtype* diff, Dtype* diff2, Dtype* diff3){
    CUDA_KERNEL_LOOP(index, nthreads){
        for (int i = 0; i < outer_num; i++)
        {
                Dtype temp_diff = top_diff[i * outer_num + index / inner_num] * (-(inner_num / 2) / ((1.0+tproduct[i * outer_num + index / inner_num])*(1.0+tproduct[ i * outer_num + index / inner_num])));
		diff[index]  += temp_diff * 2.0 * (input[index] - input[i * inner_num + index % inner_num]);
		//diff[index] += scale * temp_diff * 2.0 * (input[index] - input[i * inner_num + index % inner_num]);            
                //diff3[index] += 2.0*(input[index] - input[i * inner_num + index % inner_num]);
        }
   }
}


template <typename Dtype>
__global__ void CleanBlob(const int nthreads, Dtype* clean_blob)
{
	CUDA_KERNEL_LOOP(index, nthreads){
		clean_blob[index] = Dtype(0);
	}
}

template <typename Dtype>
void PairwiseLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* diff = temp_.mutable_gpu_data();
    Dtype* count = temp_.mutable_gpu_diff();
    Dtype* diff2 = diff2_.mutable_gpu_data();
    Dtype* diff3 = diff3_.mutable_gpu_data();
    const Dtype* similarity = pairwise_sim_.gpu_data();
    const Dtype* exp_product = loss_.gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();

    int nthreads = outer_num_ * outer_num_;
  
    //calculate diff
    BackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, outer_num_, inner_num_, similarity,
                exp_product,tsne_.gpu_data(),threshold_, count, diff);
        
    //copy to bottom_diff
    Dtype count_num;
    caffe_gpu_asum(nthreads, count, &count_num);
    if (method == 0){
      // use t-sne
      const Dtype* tproduct = tsne_.gpu_data();
      //CleanBlob<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()),CAFFE_CUDA_NUM_THREADS>>>(bottom[0]->count(), bottom[0]->mutable_gpu_diff());
      //caffe_gpu_memcpy(sizeof(Dtype)*bottom[0]->count(), bottom[0]->gpu_diff(), bottom[0]->mutable_cpu_data());
      //Dtype temp_count;
      //caffe_gpu_asum(sizeof(Dtype)*bottom[0]->count(), bottom[0]->mutable_gpu_diff(), &temp_count);
      //LOG(INFO)<<"temp_count:"<<temp_count;
      //TSNEBackward11<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      //  CAFFE_CUDA_NUM_THREADS>>>(nthreads, loss_weight_/ (count_num > 0? count_num : Dtype(1)), outer_num_, inner_num_, diff,
      //  tproduct, bottom_data, bottom_data, bottom_diff, diff2, diff3);
      //TSNEBackward12<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      //  CAFFE_CUDA_NUM_THREADS>>>(nthreads, loss_weight_/ (count_num > 0? count_num : Dtype(1)), outer_num_, inner_num_, diff,
      //  tproduct, bottom_data, bottom_data, bottom_diff, diff2, diff3);
      //TSNEBackward21<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      //  CAFFE_CUDA_NUM_THREADS>>>(nthreads, loss_weight_/ (count_num > 0? count_num : Dtype(1)), outer_num_, inner_num_, diff,
      //  tproduct, bottom_data, bottom_data, bottom_diff, diff2, diff3);
      //TSNEBackward22<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      //  CAFFE_CUDA_NUM_THREADS>>>(nthreads, loss_weight_/ (count_num > 0? count_num : Dtype(1)), outer_num_, inner_num_, diff,
      //  tproduct, bottom_data, bottom_data, bottom_diff, diff2, diff3);
	
      TSNEBackward31<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
       CAFFE_CUDA_NUM_THREADS>>>(outer_num_ * inner_num_, loss_weight_/ (count_num > 0? count_num : Dtype(1)), outer_num_, inner_num_, diff, tproduct, similarity, threshold_,  bottom_data,  bottom_diff, diff2, diff3);
      TSNEBackward32<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
       CAFFE_CUDA_NUM_THREADS>>>(outer_num_ * inner_num_, loss_weight_/ (count_num > 0? count_num : Dtype(1)), outer_num_, inner_num_, diff, tproduct, similarity, threshold_, bottom_data,  bottom_diff, diff2, diff3);

      
      //caffe_gpu_memcpy(diff2_.count() * sizeof(Dtype), diff2_.gpu_data(), diff2_.mutable_cpu_data());
      //nthreads = outer_num_ * outer_num_;
      //std::ofstream fout2("diff2out.txt", ios::out);
      //for (int i = 0; i < diff2_.count(); i++)
      //{   
       //   fout2<<diff2_.cpu_data()[i]<<' ';
      //}
      //fout2.close();
      //caffe_gpu_memcpy(diff3_.count() * sizeof(Dtype), diff3_.gpu_data(), diff3_.mutable_cpu_data());
      //fout2.open("diff3out.txt", ios::out);
      //for (int i = 0; i < diff3_.count(); i++)
      //{   
      //     fout2<<diff3_.cpu_data()[i]<<' ';
      //}
      //fout2.close();
      //for (int i = 0; i < outer_num_; i++)
	//for (int j = 0; j < inner_num_; j++)
	//	{
//			diff3_.mutable_cpu_data()[i * 48 + j] = 0;
//		}
//      fout2.open("diff3outempty.txt", ios::out);
//       for (int i = 0; i < diff3_.count(); i++)
//       {
//            fout2<<diff3_.cpu_data()[i]<<' ';
//       }
//       fout2.close();

//      caffe_gpu_memcpy(bottom[0]->count() * sizeof(Dtype), bottom[0]->gpu_data(), bottom[0]->mutable_cpu_data());
//	for (int i = 0; i < outer_num_; i++)
//         for (int j = 0; j < outer_num_; j++) 
//          for (int k = 0; k < inner_num_; k++)
//                 {
//                        diff3_.mutable_cpu_data()[i * 48 + k] +=2*( bottom[0]->cpu_data()[i * 48 + k] - bottom[0]->cpu_data()[j * 48 + k]);
 //                       diff3_.mutable_cpu_data()[j * 48 + k] +=2* (bottom[0]->cpu_data()[j * 48 + k] - bottom[0]->cpu_data()[i * 48 + k]);
 //               }
//       fout2.open("diff3out2.txt", ios::out);
//       for (int i = 0; i < diff3_.count(); i++)
//       {
//            fout2<<diff3_.cpu_data()[i]<<' ';
//       }
//       fout2.close();
//       LOG(INFO)<<"testpos0:"<<bottom[0]->cpu_data()[31 * 48 + 47] - bottom[0]->cpu_data()[31 * 48 + 47];
//       LOG(INFO)<<"testpos:"<<bottom[0]->cpu_data()[0 * 48 + 47] - bottom[0]->cpu_data()[31 * 48 + 47];
    }
    else{
      // use dhn
      caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, outer_num_, inner_num_, outer_num_, 
        loss_weight_ / (count_num > 0 ? count_num : Dtype(1)), diff, bottom_data, 
        Dtype(0.0), bottom_diff);
    }
  }
//  caffe_gpu_memcpy(sizeof(Dtype)*temp_.count(), temp_.gpu_data(), temp_.mutable_cpu_data());
//  std::ofstream fout("diffout.txt", ios::out);
//  for (int i = 0; i < temp_.count(); i++)
//  {
        //fout.write((char*)(bottom[1]->cpu_data()[i]), sizeof(bottom[1]->cpu_data()[i]));
//        fout<<temp_.cpu_data()[i]<<' ';
//   }
//   fout.close();

  
  //caffe_gpu_memcpy(bottom[0]->count() * sizeof(Dtype), bottom[0]->gpu_diff(), bottom[0]->mutable_cpu_diff());
  //std::ofstream fout("bottomdiffout.txt", ios::out);
  //for (int i = 0; i < bottom[0]->count(); i++)
 // {
 //     fout<<bottom[0]->cpu_diff()[i]<<' ';
 // }
 // fout.close();

}

INSTANTIATE_LAYER_GPU_FUNCS(PairwiseLossLayer);

}  // namespace caffe
